
#include <hip/hip_runtime.h>
#if CUDA_AVAILABLE

#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include "../../include/PhysicsEngine/Scenes/Scene.h"
#include "gpuScene.h"

__global__ void processSceneKernel(Scene* scene) {
    int i = threadIdx.x;
    if (i < scene->sceneObjects.size()) {
        // Example processing: move each object along the x-axis

    }
}

void GPUScene::runSceneOnGPU(Scene* scene) {
    Scene* dev_scene = nullptr;
    cudaMalloc((void**)&dev_scene, sizeof(Scene));
    cudaMemcpy(dev_scene, scene, sizeof(Scene), cudaMemcpyHostToDevice);

    processSceneKernel<<<1, scene->sceneObjects.size()>>>(dev_scene);

    cudaMemcpy(scene, dev_scene, sizeof(Scene), cudaMemcpyDeviceToHost);
    cudaFree(dev_scene);
}

#endif